#include "hip/hip_runtime.h"
#include "kdtree_cuda.h"
#include <algorithm>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <deque>
#include <numeric>
#include <sstream>
#include <stdexcept>

namespace cu {

    static inline void CheckCudaError(const char *msg) {
        auto err = hipGetLastError();
        if (err != hipSuccess) {
            std::stringstream ss;
            ss << "CUDA error[" << msg << "]: " << hipGetErrorString(err) << "\n";
            throw std::runtime_error(ss.str());
        }
    }

    template<typename T>
    static void CudaMalloc(T *&ptr, size_t num) {
        hipMalloc((void **) &ptr, sizeof(T) * num);
    }

    void KDTree::Clear() {
        hipFree(nodes_);
        hipFree(pts_);
        pt_num_ = 0;
    }

    static auto BuildKDtree(const std::vector <Point> &pts) {
        using Node = KDTree::Node;
        std::vector <Node> nodes;
        if (pts.empty()) {
            return nodes;
        }
        nodes.reserve(pts.size());
        std::vector<int> indices(pts.size());
        std::iota(indices.begin(), indices.end(), 0);

        struct BuildTask {
            BuildTask(Node &_node, int *_indices, int _pt_num, int _depth)
                    : node(&_node), indices(_indices), pt_num(_pt_num), depth(_depth) {
            }

            Node *node; //!< the node to be built
            int *indices; //!< the beginning pointer to the indices
            int pt_num; //!< num of points belong to the node
            int depth; //!< depth of the node
        };

        nodes.emplace_back(); // the first node is root
        std::deque <BuildTask> tasks;
        tasks.emplace_back(nodes[0], indices.data(), int(pts.size()), 0);
        while (!tasks.empty()) {
            auto crt_task = tasks.front();
            tasks.pop_front();
            const int pt_num = crt_task.pt_num;

            const int axis = crt_task.depth % 2;
            const int mid = pt_num / 2;
            int *ids = crt_task.indices;
            std::nth_element(ids, ids + mid, ids + pt_num, [&](int lhs, int rhs) {
                return pts[lhs][axis] < pts[rhs][axis];
            });
            auto &crt_node = *crt_task.node;
            crt_node.pid = ids[mid];
            crt_node.axis = axis;
            int next_depth = crt_task.depth + 1;
            auto CreateChildNodeTask = [&](int *idx_start, int pt_num) {
                nodes.emplace_back();
                int nid = static_cast<int>(nodes.size() - 1);
                tasks.emplace_back(nodes[nid], idx_start, pt_num, next_depth);
                return nid;
            };
            if (mid > 0) {
                crt_node.next[0] = CreateChildNodeTask(ids, mid);
            }
            int r_num = pt_num - mid - 1;
            if (r_num > 0) {
                crt_node.next[1] = CreateChildNodeTask(ids + mid + 1, r_num);
            }
        }
        return nodes;
    }

    void KDTree::Build(const std::vector <Point> &pts) {
        Clear();
        if (pts.empty()) {
            return;
        }
        const auto pt_num = static_cast<int>(pts.size());
        CudaMalloc(pts_, pt_num);
        CudaMalloc(nodes_, pt_num);
        CheckCudaError("KDTree::Build");
        hipMemcpyAsync(pts_, pts.data(), sizeof(Point) * pt_num, hipMemcpyHostToDevice);
        auto nodes = BuildKDtree(pts);
        hipMemcpy(nodes_, nodes.data(), sizeof(Node) * pt_num, hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        pt_num_ = pts.size();
        CheckCudaError("KDTree::Build");
    }

    __device__ float SquaredDistance(const Point &pt1, const Point &pt2) {
        float dx = pt1.coord[0] - pt2.coord[0];
        float dy = pt1.coord[1] - pt2.coord[1];
        return dx * dx + dy * dy;
    }

    struct CheckTask {
        CheckTask() = default;

        __device__ void Set(int _nid, float _diff_square) {
            nid = _nid;
            diff_square = _diff_square;
        }

        int nid{-1}; //!< node index
        float diff_square{0.0};
    };

    __device__ void
    Search(const KDTree::Node *nodes, const Point *pts, const Point &query, int &ret_idx, float &ret_dist) {
        int guess;
        float min_sd = 3e30;
        CheckTask tasks[15]; // 15 buf stack can hold about 30k nodes
        tasks[0].Set(0, 0.0f);
        int stack_size = 1;
        while (stack_size > 0) {
            // pop stack top
            const auto &crt_task = tasks[--stack_size];
            if (crt_task.diff_square >= min_sd) {
                continue;
            }
            const auto &node = nodes[crt_task.nid];
            const auto pid = node.pid;
            const auto &train = pts[pid];
            float sd = SquaredDistance(train, query);
            if (sd < min_sd) {
                guess = pid;
                min_sd = sd;
            }

            const auto axis = node.axis;
            const int dir = query.coord[axis] < train.coord[axis] ? 0 : 1;
            auto diff = query.coord[axis] - train.coord[axis];
            auto PushNode = [&](float diff_square, int nid) {
                if (nid < 0) {
                    return;
                }
                tasks[stack_size++].Set(nid, diff_square);
            };
            PushNode(diff * diff, node.next[!dir]);
            PushNode(0.0f, node.next[dir]);
        }

        ret_idx = guess;
        ret_dist = sqrt(min_sd);
    }

    __global__ void SearchBatch(const KDTree::Node *nodes, const Point *pts, const Point *queries, size_t query_num,
                                int *ret_indices, float *ret_dists) {
        auto idx = blockIdx.x * blockDim.x + threadIdx.x;
        if (idx >= query_num) {
            return;
        }
        Search(nodes, pts, queries[idx], ret_indices[idx], ret_dists[idx]);
    }

    void KDTree::NNSearchBatch(const std::vector <Point> &queries, std::vector<int> &indices,
                               std::vector<float> &min_dists) const {
        if (pt_num_ == 0 || queries.empty()) {
            return;
        }

        const size_t query_num = queries.size();
        const int thd_num = 512;
        const int blk_num = query_num / thd_num + ((query_num % thd_num) ? 1 : 0);

        // copy queries to device
        Point *gpu_queries;
        int *gpu_ret_indices;
        float *gpu_ret_dists;
        CudaMalloc(gpu_queries, query_num);
        CudaMalloc(gpu_ret_indices, query_num);
        CudaMalloc(gpu_ret_dists, query_num);
        CheckCudaError("KDTree::NNSearchBatch");
        hipMemcpy(gpu_queries, queries.data(), sizeof(Point) * query_num, hipMemcpyHostToDevice);
        CheckCudaError("KDTree::NNSearchBatch");

        // do NN search
        SearchBatch<<<blk_num, thd_num>>>(nodes_, pts_, gpu_queries, query_num,
                                          gpu_ret_indices, gpu_ret_dists);
        CheckCudaError("KDTree::NNSearchBatch");
        hipDeviceSynchronize();

        // copy data back to host
        indices.resize(query_num);
        min_dists.resize(query_num);
        hipMemcpyAsync(indices.data(), gpu_ret_indices, sizeof(int) * query_num, hipMemcpyDeviceToHost);
        hipMemcpy(min_dists.data(), gpu_ret_dists, sizeof(float) * query_num, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        hipFree(gpu_queries);
        hipFree(gpu_ret_indices);
        hipFree(gpu_ret_dists);
    }

} // namespace cu